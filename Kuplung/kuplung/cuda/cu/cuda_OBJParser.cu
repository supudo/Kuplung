//
//  cuda_OBJParser.cu
//  Kuplung
//
//  Created by Sergey Petrov on 12/3/15.
//  Copyright © 2015 supudo.net. All rights reserved.
//

#include <hip/hip_runtime.h>

#include <builtin_types.h>

extern "C"
__global__ void cudaParseOBJFile(int* obj_data,
                                 float *vertices,
                                 float *textureCoordinates,
                                 float *normals,
                                 float *indices
) {
    const unsigned int tid = threadIdx.x;
    int data = obj_data[tid];
}

void parseOBJFile(const char* obj_file_contents,
                  int obj_file_content_length,
                  float *vertices,
                  float *textureCoordinates,
                  float *normals,
                  float *indices
) {
    float *cuda_vertices, *cuda_textureCoordinates, *cuda_normals, *cuda_indices;
    const unsigned int num_threads = obj_file_content_length / 4;
    const unsigned int mem_size = sizeof(char) * obj_file_content_length;

    char *d_data;
    hipMalloc((void **) &d_data, mem_size);
    hipMemcpy(d_data, obj_file_contents, mem_size, hipMemcpyHostToDevice);

    dim3 grid(1, 1, 1);
    dim3 threads(num_threads, 1, 1);
    // execute the kernel
    cudaParseOBJFile<<< grid, threads >>>((int *) d_data, vertices, textureCoordinates, normals, indices);

    hipFree(d_data);
    hipFree(cuda_vertices);
    hipFree(cuda_textureCoordinates);
    hipFree(cuda_normals);
    hipFree(cuda_indices);
}
