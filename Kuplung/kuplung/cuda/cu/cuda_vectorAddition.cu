#include <hip/hip_runtime.h>
#include <builtin_types.h>

extern "C"
__global__ void vectorAdditionCUDA(const float* a, const float* b, float* c, int n)
{
    int ii = blockDim.x * blockIdx.x + threadIdx.x;
    if (ii < n)
        c[ii] = a[ii] + b[ii];
}

void vectorAddition(const float* a, const float* b, float* c, int n) {
    float *a_cuda, *b_cuda, *c_cuda;
    unsigned int nBytes = sizeof(float) * n;
    int threadsPerBlock = 256;
    int blocksPerGrid   = (n + threadsPerBlock - 1) / threadsPerBlock;

    // allocate and copy memory into the device
    hipMalloc((void **)& a_cuda, nBytes);
    hipMalloc((void **)& b_cuda, nBytes);
    hipMalloc((void **)& c_cuda, nBytes);
    hipMemcpy(a_cuda, a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(b_cuda, b, nBytes, hipMemcpyHostToDevice);

    vectorAdditionCUDA<<<blocksPerGrid, threadsPerBlock>>>(a_cuda, b_cuda, c_cuda, n);

    // load the answer back into the host
    hipMemcpy(c, c_cuda, nBytes, hipMemcpyDeviceToHost);

    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(c_cuda);
}
